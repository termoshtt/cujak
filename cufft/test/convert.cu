
#include "../Converter.hpp"
#include "../../thrust.hpp"
#include <fstream>
#include <iomanip>

typedef typename cujak::traits<float>::Real Real;
typedef typename cujak::traits<float>::Complex Complex;
typedef thrust::device_vector<Real> Vector;
typedef thrust::device_vector<Complex> cVector;
typedef thrust::host_vector<Real> hVector;
typedef thrust::host_vector<Complex> hcVector;

using namespace cujak::fft2d;

const int Nx = 128, Ny = 128;

int main(int argc, char const *argv[]) {
  Field<float> F(Nx, Ny);
  Coefficient<float> C(Nx, Ny);

  ConverterC2R<float> c2r(Nx, Ny);
  ConverterR2C<float> r2c(Nx, Ny);

  C.set(0, 1, cujak::complex<float>(0.0, 1.0));
  C.set(1, 0, cujak::complex<float>(0.0, 1.0));

  C.output_ascii("coef1.dat");
  c2r(C, F);
  F.output_ascii("field.dat");
  r2c(F, C);
  C.output_ascii("coef2.dat");

  return 0;
}
