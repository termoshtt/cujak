
#include "Converter.hpp"

#include "../exception.hpp"

namespace cujak{
namespace fft2d {

template <> ConverterC2R<float>::ConverterC2R(int Nx, int Ny) {
  exec(hipfftPlan2d(&plan, Nx, Ny, HIPFFT_C2R));
}
template <> ConverterC2R<double>::ConverterC2R(int Nx, int Ny) {
  exec(hipfftPlan2d(&plan, Nx, Ny, HIPFFT_Z2D));
}

template <>
void ConverterC2R<float>::operator()(const Complex *uf, Real *u) const {
  exec(hipfftExecC2R(plan, const_cast<Complex *>(uf), u));
}

template <>
void ConverterC2R<double>::operator()(const Complex *uf, Real *u) const {
  exec(hipfftExecZ2D(plan, const_cast<Complex *>(uf), u));
}

template <> ConverterR2C<float>::ConverterR2C(int Nx, int Ny) : Nx(Nx), Ny(Ny) {
  exec(hipfftPlan2d(&plan, Nx, Ny, HIPFFT_R2C));
}
template <>
ConverterR2C<double>::ConverterR2C(int Nx, int Ny)
    : Nx(Nx), Ny(Ny) {
  exec(hipfftPlan2d(&plan, Nx, Ny, HIPFFT_D2Z));
}

template <typename T> class mult {
  T a;

public:
  typedef typename traits<T>::Complex Complex;

  mult(T a) : a(a) {}

  __device__ Complex operator()(Complex c) {
    c.x *= a;
    c.y *= a;
    return c;
  }
};

template <>
void ConverterR2C<float>::operator()(const Real *u, Complex *uf) const {
  exec(hipfftExecR2C(plan, const_cast<Real *>(u), uf));
  thrust::device_ptr<Complex> p(uf);
  int stride = Ny / 2 + 1;
  Real a = 1. / (Nx * Ny);
  thrust::transform(p, p + stride, p, mult<Real>(a));
  thrust::transform(p + stride, p + Nx * stride, p + stride, mult<Real>(2 * a));
}

template <>
void ConverterR2C<double>::operator()(const Real *u, Complex *uf) const {
  exec(hipfftExecD2Z(plan, const_cast<Real *>(u), uf));
  thrust::device_ptr<Complex> p(uf);
  int stride = Ny / 2 + 1;
  Real a = 1. / (Nx * Ny);
  thrust::transform(p, p + stride, p, mult<Real>(a));
  thrust::transform(p + stride, p + Nx * stride, p + stride, mult<Real>(2 * a));
}

} // namespace fft2d
} // namespace cujak
