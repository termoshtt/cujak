#include "hip/hip_runtime.h"

#include "../operators.hpp"
#include "../math.hpp"

typedef typename cujak::traits<float>::Real Real;
typedef typename cujak::traits<float>::Complex Complex;

using namespace cujak::operators;
using namespace cujak::math;

int main(int argc, char const* argv[]) {
  Real a = 2.0;
  Complex c = {1,2};
  a + c;
  c + a;
  a - c;
  c - a;
  a * c;
  c * a;

  2 * c;
  c + c;
  c - c; 
  c * c;

  exp(c);

  return 0;
}
