
#include "../exception.hpp"
#include <iostream>

void test_exception(hipError_t err) {
  try {
    cujak::exec(err);
  }
  catch (cujak::CUDA_ERROR &e) {
    std::cout << "Error type: " << e.err << "\nMessage: "
              << e.what() << std::endl;
  }
}

void test_exception(hipfftResult err) {
  try {
    cujak::exec(err);
  }
  catch (cujak::CUFFT_ERROR &e) {
    std::cout << "Error type: " << e.err << "\nMessage: "
              << e.what() << std::endl;
  }
}


int main(int argc, char const *argv[]) {
  test_exception(hipSuccess);
  test_exception(hipErrorMissingConfiguration);
  test_exception(hipErrorOutOfMemory);
  test_exception(hipErrorNotInitialized);
  test_exception(hipErrorLaunchFailure);
  test_exception(hipErrorPriorLaunchFailure);
  test_exception(hipErrorLaunchTimeOut);
  test_exception(hipErrorLaunchOutOfResources);
  test_exception(hipErrorInvalidDeviceFunction);
  test_exception(hipErrorInvalidConfiguration);
  test_exception(hipErrorInvalidDevice);
  test_exception(hipErrorInvalidValue);
  test_exception(hipErrorInvalidPitchValue);
  test_exception(hipErrorInvalidSymbol);
  test_exception(hipErrorMapFailed);
  test_exception(hipErrorUnmapFailed);
  test_exception(cudaErrorInvalidHostPointer);
  test_exception(hipErrorInvalidDevicePointer);
  test_exception(hipErrorInvalidTexture);
  test_exception(cudaErrorInvalidTextureBinding);
  test_exception(hipErrorInvalidChannelDescriptor);
  test_exception(hipErrorInvalidMemcpyDirection);
  test_exception(cudaErrorAddressOfConstant);
  test_exception(cudaErrorTextureFetchFailed);
  test_exception(cudaErrorTextureNotBound);
  test_exception(cudaErrorSynchronizationError);
  test_exception(cudaErrorInvalidFilterSetting);
  test_exception(cudaErrorInvalidNormSetting);
  test_exception(cudaErrorMixedDeviceExecution);
  test_exception(hipErrorDeinitialized);
  test_exception(hipErrorUnknown);
  test_exception(cudaErrorNotYetImplemented);
  test_exception(cudaErrorMemoryValueTooLarge);
  test_exception(hipErrorInvalidHandle);
  test_exception(hipErrorNotReady);
  test_exception(hipErrorInsufficientDriver);
  test_exception(hipErrorSetOnActiveProcess);
  test_exception(cudaErrorInvalidSurface);
  test_exception(hipErrorNoDevice);
  test_exception(hipErrorECCNotCorrectable);
  test_exception(hipErrorSharedObjectSymbolNotFound);
  test_exception(hipErrorSharedObjectInitFailed);
  test_exception(hipErrorUnsupportedLimit);
  test_exception(cudaErrorDuplicateVariableName);
  test_exception(cudaErrorDuplicateTextureName);
  test_exception(cudaErrorDuplicateSurfaceName);
  test_exception(cudaErrorDevicesUnavailable);
  test_exception(hipErrorInvalidImage);
  test_exception(hipErrorNoBinaryForGpu);
  test_exception(cudaErrorIncompatibleDriverContext);
  test_exception(hipErrorPeerAccessAlreadyEnabled);
  test_exception(hipErrorPeerAccessNotEnabled);
  test_exception(hipErrorContextAlreadyInUse);
  test_exception(hipErrorProfilerDisabled);
  test_exception(hipErrorProfilerNotInitialized);
  test_exception(hipErrorProfilerAlreadyStarted);
  test_exception(hipErrorProfilerAlreadyStopped);
  test_exception(hipErrorAssert);
  test_exception(cudaErrorTooManyPeers);
  test_exception(hipErrorHostMemoryAlreadyRegistered);
  test_exception(hipErrorHostMemoryNotRegistered);
  test_exception(hipErrorOperatingSystem);
  test_exception(hipErrorPeerAccessUnsupported);
  test_exception(cudaErrorLaunchMaxDepthExceeded);
  test_exception(cudaErrorLaunchFileScopedTex);
  test_exception(cudaErrorLaunchFileScopedSurf);
  test_exception(cudaErrorSyncDepthExceeded);
  test_exception(cudaErrorLaunchPendingCountExceeded);
  test_exception(cudaErrorNotPermitted);
  test_exception(hipErrorNotSupported);
  test_exception(cudaErrorHardwareStackError);
  test_exception(cudaErrorIllegalInstruction);
  test_exception(cudaErrorMisalignedAddress);
  test_exception(cudaErrorInvalidAddressSpace);
  test_exception(cudaErrorInvalidPc);
  test_exception(hipErrorIllegalAddress);
  test_exception(cudaErrorStartupFailure);
  test_exception(cudaErrorApiFailureBase);

  test_exception(HIPFFT_SUCCESS);
  test_exception(HIPFFT_INVALID_PLAN);
  test_exception(HIPFFT_ALLOC_FAILED);
  test_exception(HIPFFT_INVALID_TYPE);
  test_exception(HIPFFT_INVALID_VALUE);
  test_exception(HIPFFT_INTERNAL_ERROR);
  test_exception(HIPFFT_EXEC_FAILED);
  test_exception(HIPFFT_SETUP_FAILED);
  test_exception(HIPFFT_INVALID_SIZE);
  test_exception(HIPFFT_UNALIGNED_DATA);
  test_exception(HIPFFT_INCOMPLETE_PARAMETER_LIST);
  test_exception(HIPFFT_INVALID_DEVICE);
  test_exception(HIPFFT_PARSE_ERROR);
  test_exception(HIPFFT_NO_WORKSPACE);
  return 0;
}
