#include "hip/hip_runtime.h"

#include "../cufft.hpp"
#include "../thrust.hpp"

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

namespace FFT = cujak::fft2d;

typedef typename FFT::traits<float>::Real Real;
typedef typename FFT::traits<float>::Complex Complex;

using cujak::raw_pointer;

int main(int argc, char const *argv[]) {
  const int Nx = 16;
  const int Ny = 16;

  const int N = Nx * Ny;
  const int Nc = FFT::padded_size(Nx, Ny);

  thrust::host_vector<Real> u_host(N);
  for (int i = 0; i < Nx; ++i) {
    for (int j = 0; j < Ny; ++j) {
      double x = ((double)i) / Nx;
      double y = ((double)j) / Ny;
      u_host[Ny * i + j] = cos(2 * M_PI * x) * cos(2 * M_PI * y);
    }
  }

  thrust::device_vector<Real> u = u_host;
  thrust::device_vector<Complex> uf(Nc);

  FFT::r2c<float>(Nx, Ny, raw_pointer(u), raw_pointer(uf));
  FFT::c2r<float>(Nx, Ny, raw_pointer(uf), raw_pointer(u));
  return 0;
}
