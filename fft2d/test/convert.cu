#include "hip/hip_runtime.h"

#include "../plan.hpp"
#include "../algorithm.hpp"
#include "../pb.hpp"
#include "../../thrust.hpp"

#include <gtest/gtest.h>

namespace FFT = cujak::fft2d;

class ConvertTest : public ::testing::Test {
protected:
  ConvertTest() : F(Nx, Ny), C(Nx, Ny), c2r(Nx, Ny), r2c(Nx, Ny) {}
  const int Nx = 128;
  const int Ny = 128;
  const double Lx = 7.2354234;
  const double Ly = 3.2345234;
  const double kx = 2 * M_PI / Lx;
  const double ky = 2 * M_PI / Ly;

  const double eps = 1e-4;

  FFT::Field<float> F;
  FFT::Coefficient<float> C;
  FFT::planC2R<float> c2r;
  FFT::planR2C<float> r2c;

  virtual void SetUp() {
    F.property.set_lx(Lx);
    F.property.set_ly(Ly);
    C.property.set_lx(Lx);
    C.property.set_ly(Ly);
  }
};

TEST_F(ConvertTest, r2c2r) {
  fill_random(F);
  FFT::Field<float> ans(Nx, Ny);
  thrust::copy(F.begin(), F.end(), ans.begin());
  r2c(F, C);
  c2r(C, F);
  for (int i = 0; i < Nx; ++i) {
    for (int j = 0; j < Ny; ++j) {
      EXPECT_NEAR(ans(i, j), F(i, j), eps);
    }
  }
}

TEST_F(ConvertTest, c2r2c) {
  fill_random(C);
  FFT::Coefficient<float> ans(Nx, Ny);
  thrust::copy(C.begin(), C.end(), ans.begin());
  c2r(C, F);
  r2c(F, C);
  for (int i = 0; i < Nx; ++i) {
    for (int j = 0; j < cujak::fft2d::calc_stride(Ny); ++j) {
      EXPECT_NEAR(ans(i, j).x, C(i, j).x, eps);
      EXPECT_NEAR(ans(i, j).y, C(i, j).y, eps);
    }
  }
}

TEST_F(ConvertTest, manualR2C) {
  set(F, [=](double x, double y) {
    return 1 + sin(kx * x) + sin(ky * y) + cos(kx * x + 2 * ky * y) +
           sin(2 * kx * x + 3 * ky * y) + cos(kx * x - 3 * ky * y) +
           cos(3 * kx * x - 5 * ky * y);
  });
  r2c(F, C);
  for (int i = -Nx / 2; i < Nx / 2; i++) {
    for (int j = 0; j < FFT::calc_stride(Ny); ++j) {
      auto c = C(i, j);
      if (i == 0 && j == 0) { // 1
        EXPECT_NEAR(1.0, c.x, eps) << i << " " << j;
        EXPECT_NEAR(0.0, c.y, eps) << i << " " << j;
        continue;
      }
      if (i == 0 && j == 1) { // sin(y)
        EXPECT_NEAR(0.0, c.x, eps) << i << " " << j;
        EXPECT_NEAR(-0.5, c.y, eps) << i << " " << j;
        continue;
      }
      if (i == 1 && j == 0) { // sin(x)
        EXPECT_NEAR(0.0, c.x, eps) << i << " " << j;
        EXPECT_NEAR(-0.5, c.y, eps) << i << " " << j;
        continue;
      }
      if (i == -1 && j == 0) { // sin(x): 2つめ
        EXPECT_NEAR(0.0, c.x, eps) << i << " " << j;
        EXPECT_NEAR(0.5, c.y, eps) << i << " " << j;
        continue;
      }
      if (i == 1 && j == 2) { // cos(x + 2y)
        EXPECT_NEAR(0.5, c.x, eps) << i << " " << j;
        EXPECT_NEAR(0.0, c.y, eps) << i << " " << j;
        continue;
      }
      if (i == 2 && j == 3) { // sin(2x + 3y)
        EXPECT_NEAR(0.0, c.x, eps) << i << " " << j;
        EXPECT_NEAR(-0.5, c.y, eps) << i << " " << j;
        continue;
      }
      if (i == -1 && j == 3) { // cos(x - 3y)
        EXPECT_NEAR(0.5, c.x, eps) << i << " " << j;
        EXPECT_NEAR(0.0, c.y, eps) << i << " " << j;
        continue;
      }
      if (i == -3 && j == 5) { // cos(3x - 5y)
        EXPECT_NEAR(0.5, c.x, eps) << i << " " << j;
        EXPECT_NEAR(0.0, c.y, eps) << i << " " << j;
        continue;
      }
      EXPECT_NEAR(0.0, c.x, eps) << i << " " << j;
      EXPECT_NEAR(0.0, c.y, eps) << i << " " << j;
    }
  }
}
