#include "hip/hip_runtime.h"

#include "../plan.hpp"
#include "../algorithm.hpp"
#include "../pb.hpp"
#include "../../thrust.hpp"

namespace FFT = cujak::fft2d;

const int Nx = 256, Ny = 128;

int main(int argc, char const *argv[]) {
  FFT::Field<float> F(Nx, Ny);
  FFT::Coefficient<float> C(Nx, Ny);

  FFT::planC2R<float> c2r(Nx, Ny);
  FFT::planR2C<float> r2c(Nx, Ny);

  F.property.set_lx(4 * M_PI);
  F.property.set_ly(2 * M_PI);

  set(F, [](double x, double y) {
    return 1 + sin(x) + sin(y) + cos(x + 2 * y) + sin(2 * x + 3 * y) +
           cos(x - 3 * y) + cos(3 * x - 5 * y);
  });
  save_pb(F, "pre");
  r2c(F, C);
  for (int i = -Nx / 2; i < Nx / 2; i++) {
    for (int j = 0; j < FFT::calc_stride(Ny); ++j) {
      auto c = C(i, j);
      if (abs(c.x) > 0.1) {
        std::cout << "i,j = " << i << ", " << j << ", c.x = " << c.x
                  << std::endl;
      }
      if (abs(c.y) > 0.1) {
        std::cout << "i,j = " << i << ", " << j << ", c.y = " << c.y
                  << std::endl;
      }
    }
  }
  save_pb(C, "inter");
  c2r(C, F);
  save_pb(F, "post");

  return 0;
}
