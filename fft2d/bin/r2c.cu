
#include "../pb.hpp"
#include "../plan.hpp"

namespace FFT = cujak::fft2d;

template <typename T> void work(std::string filename) {
  auto pb = FFT::load_field<T>(filename);
  FFT::Field<T> F = FFT::pb2field<T>(pb);
  const int Nx = F.size_x();
  const int Ny = F.size_y();
  FFT::planR2C<T> p(Nx, Ny);
  FFT::Coefficient<T> C(Nx, Ny);
  C.property = F.property;
  p(F, C);
  std::string prefix(filename.begin(), filename.end() - 3);
  FFT::save_pb(C, prefix);
}

int main(int argc, char const *argv[]) {
  if (argc != 2) {
    std::cout << "Usage: " << argv[0] << " [filename]" << std::endl;
    return 1;
  }
  std::string filename(argv[1]);
  std::string ext(filename.end() - 3, filename.end());
  if (ext == FFT::field_ext<float>()) {
    work<float>(filename);
    return 0;
  }
  if (ext == FFT::field_ext<double>()) {
    work<double>(filename);
    return 0;
  }
  std::cerr << "Invalid filename: " + filename << std::endl;
  return 1;
}
