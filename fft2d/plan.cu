
#include "plan.hpp"

#include "../exception.hpp"

namespace cujak {
namespace fft2d {

template <> planC2R<float>::planC2R(int Nx, int Ny) {
  exec(hipfftPlan2d(&plan, Nx, Ny, HIPFFT_C2R));
}
template <> planC2R<double>::planC2R(int Nx, int Ny) {
  exec(hipfftPlan2d(&plan, Nx, Ny, HIPFFT_Z2D));
}

template <> void planC2R<float>::operator()(const Complex *uf, Real *u) const {
  exec(hipfftExecC2R(plan, const_cast<Complex *>(uf), u));
}

template <> void planC2R<double>::operator()(const Complex *uf, Real *u) const {
  exec(hipfftExecZ2D(plan, const_cast<Complex *>(uf), u));
}

template <> planR2C<float>::planR2C(int Nx, int Ny) : Nx(Nx), Ny(Ny) {
  exec(hipfftPlan2d(&plan, Nx, Ny, HIPFFT_R2C));
}
template <> planR2C<double>::planR2C(int Nx, int Ny) : Nx(Nx), Ny(Ny) {
  exec(hipfftPlan2d(&plan, Nx, Ny, HIPFFT_D2Z));
}

template <typename T> class mult {
  T a;

public:
  typedef typename traits<T>::Complex Complex;

  mult(T a) : a(a) {}

  __device__ Complex operator()(Complex c) {
    c.x *= a;
    c.y *= a;
    return c;
  }
};

template <> void planR2C<float>::operator()(const Real *u, Complex *uf) const {
  exec(hipfftExecR2C(plan, const_cast<Real *>(u), uf));
  thrust::device_ptr<Complex> p(uf);
  int stride = Ny / 2 + 1;
  Real a = 1. / (Nx * Ny);
  thrust::transform(p, p + stride, p, mult<Real>(a));
  thrust::transform(p + stride, p + Nx * stride, p + stride, mult<Real>(2 * a));
}

template <> void planR2C<double>::operator()(const Real *u, Complex *uf) const {
  exec(hipfftExecD2Z(plan, const_cast<Real *>(u), uf));
  thrust::device_ptr<Complex> p(uf);
  int stride = Ny / 2 + 1;
  Real a = 1. / (Nx * Ny);
  thrust::transform(p, p + stride, p, mult<Real>(a));
  thrust::transform(p + stride, p + Nx * stride, p + stride, mult<Real>(2 * a));
}

} // namespace fft2d
} // namespace cujak
