#include "hip/hip_runtime.h"

#include "pb.hpp"
#include <fstream>

namespace cujak {
namespace fft2d {

template <> std::string field_ext<float>() { return ".ff"; }
template <> std::string field_ext<double>() { return ".df"; }
template <> std::string coef_ext<float>() { return ".fc"; }
template <> std::string coef_ext<double>() { return ".dc"; }

inline std::string add_ext(std::string filename, std::string ext) {
  auto found = filename.find(ext);
  if (found == (filename.size() - ext.size())) {
    return filename;
  }
  return filename + ext;
}

template <typename PB> inline void pb2ofs(PB &pb, std::string filename) {
  std::ofstream ofs(filename,
                    std::ios::out | std::ios::binary | std::ios::trunc);
  if (!ofs)
    throw std::runtime_error("Cannot open file: " + filename);
  pb.SerializeToOstream(&ofs);
}

template <class PB> PB load_pb(std::string filename) {
  std::ifstream ifs(filename, std::ios::in | std::ios::binary);
  if (!ifs)
    throw std::runtime_error("Cannot open file: " + filename);
  PB pb;
  pb.ParseFromIstream(&ifs);
  return pb;
}

template <typename T> pb_Field<T> load_field(std::string filename) {
  return load_pb<pb_Field<T> >(filename);
}
template pb_Field<float> load_field<float>(std::string);
template pb_Field<double> load_field<double>(std::string);

template <typename T> pb_Coefficient<T> load_coef(std::string filename) {
  return load_pb<pb_Coefficient<T> >(filename);
}
template pb_Coefficient<float> load_coef<float>(std::string);
template pb_Coefficient<double> load_coef<double>(std::string);

template <typename T> void save_pb(Field_wrapper<T> &F, std::string filename) {
  const int Nx = F.size_x();
  const int Ny = F.size_y();
  typename pb_traits<T>::Field pb_F;
  pb_F.set_nx(Nx);
  pb_F.set_ny(Ny);
  *(pb_F.mutable_property()) = F.property;

  thrust::host_vector<T> data = F.data();
  for (int i = 0; i < Nx; ++i) {
    for (int j = 0; j < Ny; ++j) {
      pb_F.add_value(data[Ny * i + j]);
    }
  }
  filename = add_ext(filename, field_ext<T>());
  pb2ofs(pb_F, filename);
}
template void save_pb(Field_wrapper<float> &F, std::string filename);
template void save_pb(Field_wrapper<double> &F, std::string filename);

template <typename T>
void save_pb(Coefficient_wrapper<T> &C, std::string filename) {
  const int Nx = C.size_x();
  const int Ny = C.size_y();
  const int stride = C.get_stride();
  typename pb_traits<T>::Coefficient pb_C;
  pb_C.set_nx(Nx);
  pb_C.set_ny(Ny);
  for (int i = 0; i < Nx; ++i) {
    for (int j = 0; j < stride; ++j) {
      auto c = C(i, j);
      typename pb_traits<T>::Complex pb_c;
      pb_c.set_real(c.x);
      pb_c.set_imag(c.y);
      *(pb_C.add_value()) = pb_c;
    }
  }
  filename = add_ext(filename, coef_ext<T>());
  pb2ofs(pb_C, filename);
}
template void save_pb(Coefficient_wrapper<float> &C, std::string filename);
template void save_pb(Coefficient_wrapper<double> &C, std::string filename);

} // namespace fft2d
} // namespace cujak
