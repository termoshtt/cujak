
#include "algorithm.hpp"
#include <hiprand.h>

namespace cujak {
namespace fft2d {

template <typename T>
void set(Field_wrapper<T> &F, std::function<double(double, double)> f) {
  const int Nx = F.size_x();
  const int Ny = F.size_y();
  if (!(F.property.has_lx() && F.property.has_ly())) {
    throw std::runtime_error("Lx or Ly is not set");
  }
  double Lx = F.property.lx();
  double Ly = F.property.ly();
  rhVector<T> host(Nx * Ny);
  for (int i = 0; i < Nx; ++i) {
    for (int j = 0; j < Ny; ++j) {
      double x = i * Lx / Nx;
      double y = j * Ly / Ny;
      host[Ny * i + j] = f(x, y);
    }
  }
  F.data() = host;
}
template void set<double>(Field_wrapper<double> &F,
                          std::function<double(double, double)> f);
template void set<float>(Field_wrapper<float> &F,
                         std::function<double(double, double)> f);

template <> void fill_random(Field_wrapper<float> &F, uint64_t seed) {
  hiprandGenerator_t qrng;
  hiprandCreateGenerator(&qrng, HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetPseudoRandomGeneratorSeed(qrng, seed);
  hiprandGenerateUniform(qrng, F.get(), F.size());
  hiprandDestroyGenerator(qrng);
}
template <> void fill_random(Field_wrapper<double> &F, uint64_t seed) {
  hiprandGenerator_t qrng;
  hiprandCreateGenerator(&qrng, HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetPseudoRandomGeneratorSeed(qrng, seed);
  hiprandGenerateUniformDouble(qrng, F.get(), F.size());
  hiprandDestroyGenerator(qrng);
}

} // namespace fft2d
} // namespace cujak
