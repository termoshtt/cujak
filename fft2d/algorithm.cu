
#include "algorithm.hpp"
#include "plan.hpp"
#include <hiprand.h>

namespace cujak {
namespace fft2d {

template <typename T>
void set(Field_wrapper<T> &F, std::function<double(double, double)> f) {
  const int Nx = F.size_x();
  const int Ny = F.size_y();
  if (!(F.property.has_lx() && F.property.has_ly())) {
    throw std::runtime_error("Lx or Ly is not set");
  }
  double Lx = F.property.lx();
  double Ly = F.property.ly();
  rhVector<T> host(Nx * Ny);
  for (int i = 0; i < Nx; ++i) {
    for (int j = 0; j < Ny; ++j) {
      double x = i * Lx / Nx;
      double y = j * Ly / Ny;
      host[Ny * i + j] = f(x, y);
    }
  }
  F.data() = host;
}
template void set<double>(Field_wrapper<double> &F,
                          std::function<double(double, double)> f);
template void set<float>(Field_wrapper<float> &F,
                         std::function<double(double, double)> f);

template <> void fill_random(Field_wrapper<float> &F, uint64_t seed) {
  hiprandGenerator_t qrng;
  hiprandCreateGenerator(&qrng, HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetPseudoRandomGeneratorSeed(qrng, seed);
  hiprandGenerateUniform(qrng, F.get(), F.size());
  hiprandDestroyGenerator(qrng);
}
template <> void fill_random(Field_wrapper<double> &F, uint64_t seed) {
  hiprandGenerator_t qrng;
  hiprandCreateGenerator(&qrng, HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetPseudoRandomGeneratorSeed(qrng, seed);
  hiprandGenerateUniformDouble(qrng, F.get(), F.size());
  hiprandDestroyGenerator(qrng);
}

template <typename T>
void fill_random(Coefficient_wrapper<T> &C, uint64_t seed) {
  const int Nx = C.size_x();
  const int Ny = C.size_y();
  Field<T> F(Nx, Ny);
  fill_random(F, seed);
  r2c<T>(F, C);
}
template void fill_random<float>(Coefficient_wrapper<float> &, uint64_t);
template void fill_random<double>(Coefficient_wrapper<double> &, uint64_t);

} // namespace fft2d
} // namespace cujak
