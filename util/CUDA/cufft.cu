
#include "cufft.hpp"

namespace cujak {
namespace util {

namespace fft2d {

template <typename T>
__device__ void c2r(unsigned int Nx, unsigned int Ny,
                    const typename traits<T>::Complex *uf,
                    typename traits<T>::Real *u) {
  hipfftHandle plan;
  CUFFT_EXEC(hipfftPlan2d(&plan, Nx, Ny, HIPFFT_C2R));
  CUFFT_EXEC(hipfftExecC2R(plan, uf, u));
  CUFFT_EXEC(hipfftDestroy(plan));
}

template <typename T>
__device__ void r2c(unsigned int Nx, unsigned int Ny,
                    const typename traits<T>::Real *u,
                    typename traits<T>::Complex *uf) {
  hipfftHandle plan;
  CUFFT_EXEC(hipfftPlan2d(&plan, Nx, Ny, HIPFFT_R2C));
  CUFFT_EXEC(hipfftExecR2C(plan, u, uf));
  CUFFT_EXEC(hipfftDestroy(plan));
}

} // namespace fft2d

} // namespace util
} // namespace cujak

