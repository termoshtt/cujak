
#include "../exception.hpp"
#include <iostream>

namespace U = cujak::util;

void test_exception(hipError_t err) {
  try {
    CUDA_EXEC(err);
  }
  catch (U::CUDA_ERROR &e) {
    std::cout << "Error type: " << e.err << '\n' << "Error occured at "
              << e.filename << ":" << e.line << ":" << e.funcname << '\n'
              << e.what() << '\n' << std::endl;
  }
}

int main(int argc, char const *argv[]) {
  test_exception(hipSuccess);
  test_exception(hipErrorMissingConfiguration);
  test_exception(hipErrorOutOfMemory);
  test_exception(hipErrorNotInitialized);
  test_exception(hipErrorLaunchFailure);
  test_exception(hipErrorPriorLaunchFailure);
  test_exception(hipErrorLaunchTimeOut);
  test_exception(hipErrorLaunchOutOfResources);
  test_exception(hipErrorInvalidDeviceFunction);
  test_exception(hipErrorInvalidConfiguration);
  test_exception(hipErrorInvalidDevice);
  test_exception(hipErrorInvalidValue);
  test_exception(hipErrorInvalidPitchValue);
  test_exception(hipErrorInvalidSymbol);
  test_exception(hipErrorMapFailed);
  test_exception(hipErrorUnmapFailed);
  test_exception(cudaErrorInvalidHostPointer);
  test_exception(hipErrorInvalidDevicePointer);
  test_exception(hipErrorInvalidTexture);
  test_exception(cudaErrorInvalidTextureBinding);
  test_exception(hipErrorInvalidChannelDescriptor);
  test_exception(hipErrorInvalidMemcpyDirection);
  test_exception(cudaErrorAddressOfConstant);
  test_exception(cudaErrorTextureFetchFailed);
  test_exception(cudaErrorTextureNotBound);
  test_exception(cudaErrorSynchronizationError);
  test_exception(cudaErrorInvalidFilterSetting);
  test_exception(cudaErrorInvalidNormSetting);
  test_exception(cudaErrorMixedDeviceExecution);
  test_exception(hipErrorDeinitialized);
  test_exception(hipErrorUnknown);
  test_exception(cudaErrorNotYetImplemented);
  test_exception(cudaErrorMemoryValueTooLarge);
  test_exception(hipErrorInvalidHandle);
  test_exception(hipErrorNotReady);
  test_exception(hipErrorInsufficientDriver);
  test_exception(hipErrorSetOnActiveProcess);
  test_exception(cudaErrorInvalidSurface);
  test_exception(hipErrorNoDevice);
  test_exception(hipErrorECCNotCorrectable);
  test_exception(hipErrorSharedObjectSymbolNotFound);
  test_exception(hipErrorSharedObjectInitFailed);
  test_exception(hipErrorUnsupportedLimit);
  test_exception(cudaErrorDuplicateVariableName);
  test_exception(cudaErrorDuplicateTextureName);
  test_exception(cudaErrorDuplicateSurfaceName);
  test_exception(cudaErrorDevicesUnavailable);
  test_exception(hipErrorInvalidImage);
  test_exception(hipErrorNoBinaryForGpu);
  test_exception(cudaErrorIncompatibleDriverContext);
  test_exception(hipErrorPeerAccessAlreadyEnabled);
  test_exception(hipErrorPeerAccessNotEnabled);
  test_exception(hipErrorContextAlreadyInUse);
  test_exception(hipErrorProfilerDisabled);
  test_exception(hipErrorProfilerNotInitialized);
  test_exception(hipErrorProfilerAlreadyStarted);
  test_exception(hipErrorProfilerAlreadyStopped);
  test_exception(hipErrorAssert);
  test_exception(cudaErrorTooManyPeers);
  test_exception(hipErrorHostMemoryAlreadyRegistered);
  test_exception(hipErrorHostMemoryNotRegistered);
  test_exception(hipErrorOperatingSystem);
  test_exception(hipErrorPeerAccessUnsupported);
  test_exception(cudaErrorLaunchMaxDepthExceeded);
  test_exception(cudaErrorLaunchFileScopedTex);
  test_exception(cudaErrorLaunchFileScopedSurf);
  test_exception(cudaErrorSyncDepthExceeded);
  test_exception(cudaErrorLaunchPendingCountExceeded);
  test_exception(cudaErrorNotPermitted);
  test_exception(hipErrorNotSupported);
  test_exception(cudaErrorHardwareStackError);
  test_exception(cudaErrorIllegalInstruction);
  test_exception(cudaErrorMisalignedAddress);
  test_exception(cudaErrorInvalidAddressSpace);
  test_exception(cudaErrorInvalidPc);
  test_exception(hipErrorIllegalAddress);
  test_exception(cudaErrorStartupFailure);
  test_exception(cudaErrorApiFailureBase);

  return 0;
}
